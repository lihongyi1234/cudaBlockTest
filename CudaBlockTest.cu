#include "hip/hip_runtime.h"
#pragma once
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <winsock.h>
#include <time.h>

//#define Grid2DBlock2D
//#define Grid2DBlock1D
#define Grid2DTRANSPOSE

void initialFloat(float* ip, int size) {
	for (int i = 0; i < size; i++) {
		ip[i] = i;
	}
}

void sumMatrixOnHost(float* A, float* B, float* C, const int nx, const int ny) 
{
	float* ia = A;
	float* ib = B;
	float* ic = C;
	for (int iy = 0; iy < ny; iy++) {
		for (int ix = 0; ix < nx; ix++) {
			ic[ix] = ia[ix] + ib[ix];
		}
		ia += nx;
		ib += nx;
		ic += nx;
	}
}

__global__ void sumMatrixOnGPU2D(float* MatA, float* MatB, float* MatC, int nx, int ny)
{
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned int idx = iy * nx + ix;
	if (ix < nx && iy < ny) {
		MatC[idx] = MatA[idx] + MatB[idx];
	}
}

__global__ void transposeDiagonalRow(float* MatA, float* MatC, int nx, int ny)
{
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
	if (ix < nx && iy < ny) {
		MatC[ix * ny + iy] = MatA[iy * nx + ix];
	}
}

__global__ void transposeDiagonalCol(float* MatA, float* MatC, int nx, int ny)
{
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
	if (ix < nx && iy < ny) {
		MatC[iy * nx + ix] = MatA[ix * ny + iy];
	}
}


__global__ void sumMatrixOnGPUMix(float* MatA, float* MatB, float* MatC, int nx, int ny)
{
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int iy = blockIdx.y;
	unsigned int idx = iy * nx + ix;
	if (ix < nx && iy < ny) {
		MatC[idx] = MatA[idx] + MatB[idx];
	}
}

int main()
{

	int iDev = 0;
	hipDeviceProp_t iProp;
	hipGetDeviceProperties(&iProp, iDev);

	int nx = 1 << 14;
	int ny = 1 << 14;
	int nxy = nx * ny;
	int nBytes = nx * ny * sizeof(float);
	printf("matrix nx:%d ny:%d\n", nx, ny);
	float* h_a, * h_b, * hostRef, * gpuRef;
	h_a = new float[nxy];
	h_b = new float[nxy];
	hostRef = new float[nxy];
	gpuRef = new float[nxy];

	initialFloat(h_a, nxy);
	initialFloat(h_b, nxy);

	sumMatrixOnHost(h_a, h_b, hostRef, nx, ny);

	//malloc device global memory
	float* d_MatA;
	float* d_MatB;
	float* d_MatC;
	hipMalloc((void**)&d_MatA, nBytes);
	hipMalloc((void**)&d_MatB, nBytes);
	hipMalloc((void**)&d_MatC, nBytes);

	//transfer data from host to device
	hipMemcpy((void*)d_MatA, (void*)h_a, nBytes, hipMemcpyHostToDevice);
	hipMemcpy((void*)d_MatB, (void*)h_b, nBytes, hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	float elapsedTime = 0.f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventSynchronize(start);
	hipEventRecord(start, 0);
#ifdef Grid2DBlock2D
	//invoke kernel at host side
	int dimx = 32;
	int dimy = 32;
	dim3 threadsPerBlock(dimx, dimy);
	dim3 numBlocks((nx + dimx - 1) / threadsPerBlock.x, (ny + dimy - 1) / threadsPerBlock.y);
	sumMatrixOnGPU2D << < numBlocks, threadsPerBlock >> > (d_MatA, d_MatB, d_MatC, nx, ny);
#endif
#ifdef Grid2DBlock1D
	int dimx = 256;
	dim3 threadsPerBlock(dimx);
	dim3 numBlocks((nx + dimx - 1) / threadsPerBlock.x, ny);
	sumMatrixOnGPUMix<<<numBlocks, threadsPerBlock >>>(d_MatA, d_MatB, d_MatC, nx, ny);
#endif
#ifdef Grid2DTRANSPOSE
	int dimx = 32;
	int dimy = 32;
	dim3 threadsPerBlock(dimx, dimy);
	dim3 numBlocks((nx + dimx - 1) / threadsPerBlock.x, (ny + dimy - 1) / threadsPerBlock.y);
	//transposeDiagonalRow << < numBlocks, threadsPerBlock >> > (d_MatA, d_MatC, nx, ny);
	transposeDiagonalCol << < numBlocks, threadsPerBlock >> > (d_MatA, d_MatC, nx, ny);

#endif

	hipError_t error = hipEventRecord(stop, 0);
	error = hipEventSynchronize(stop);
	error = hipEventElapsedTime(&elapsedTime, start, stop);
	printf("sumMatrixOnGPU<<<(%d %d),(%d %d)>>> time speed:%f ms\n", numBlocks.x, numBlocks.y, threadsPerBlock.x, threadsPerBlock.y,  elapsedTime);

	hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost);

	hipFree(d_MatA);
	hipFree(d_MatB);
	hipFree(d_MatC);

	delete[]h_a;
	delete[]h_b;
	delete[]hostRef;
	delete[]gpuRef;

	return 0;
}

